#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *//*
 */

/** @file   common.cu
 *  @author Thomas Müller and Nikolaus Binder, NVIDIA
 *  @brief  Common utilities that are needed by pretty much every component of this framework.
 */

#include <tiny-cuda-nn/common.h>
#include <tiny-cuda-nn/gpu_memory.h>
#include <tiny-cuda-nn/multi_stream.h>

#include <hip/hip_runtime.h>

#include <algorithm>
#include <cctype>
#include <unordered_map>

TCNN_NAMESPACE_BEGIN

static_assert(
	__CUDACC_VER_MAJOR__ > 10 || (__CUDACC_VER_MAJOR__ == 10 && __CUDACC_VER_MINOR__ >= 2),
	"tiny-cuda-nn requires at least CUDA 10.2"
);

int cuda_device() {
	int device;
	CUDA_CHECK_THROW(hipGetDevice(&device));
	return device;
}

void set_cuda_device(int device) {
	CUDA_CHECK_THROW(hipSetDevice(device));
}

int cuda_device_count() {
	int device_count;
	CUDA_CHECK_THROW(hipGetDeviceCount(&device_count));
	return device_count;
}

bool cuda_supports_virtual_memory(int device) {
	int supports_vmm;
	CU_CHECK_THROW(hipDeviceGetAttribute(&supports_vmm, CU_DEVICE_ATTRIBUTE_VIRTUAL_ADDRESS_MANAGEMENT_SUPPORTED, device));
	return supports_vmm != 0;
}

uint32_t cuda_compute_capability(int device) {
	hipDeviceProp_t props;
	CUDA_CHECK_THROW(hipGetDeviceProperties(&props, device));
	return props.major * 10 + props.minor;
}

size_t cuda_memory_granularity(int device) {
	size_t granularity;
	hipMemAllocationProp prop = {};
	prop.type = hipMemAllocationTypePinned;
	prop.location.type = hipMemLocationTypeDevice;
	prop.location.id = 0;
	hipError_t granularity_result = hipMemGetAllocationGranularity(&granularity, &prop, hipMemAllocationGranularityMinimum);
	if (granularity_result == hipErrorNotSupported) {
		return 1;
	}
	CU_CHECK_THROW(granularity_result);
	return granularity;
}

MemoryInfo cuda_memory_info() {
	MemoryInfo info;
	CUDA_CHECK_THROW(hipMemGetInfo(&info.free, &info.total));
	info.used = info.total - info.free;
	return info;
}

std::string to_lower(std::string str) {
	std::transform(std::begin(str), std::end(str), std::begin(str), [](unsigned char c) { return (char)std::tolower(c); });
	return str;
}

std::string to_upper(std::string str) {
	std::transform(std::begin(str), std::end(str), std::begin(str), [](unsigned char c) { return (char)std::toupper(c); });
	return str;
}

template <>
std::string type_to_string<float>() {
	return "float";
}

template <>
std::string type_to_string<__half>() {
	return "__half";
}

struct StreamOwnedObjects {
	std::unordered_map<hipStream_t, std::shared_ptr<GPUMemoryArena>> stream_gpu_memory_arenas;
	std::unordered_map<int, std::shared_ptr<GPUMemoryArena>> global_gpu_memory_arenas;

	struct MultiStreamWrapper {
		~MultiStreamWrapper() {
			// Avoids free_multi_stream being called in the middle of multi_streams's destruction.
			// Note, that we don't use `.clear()` intentionally, because it would also have issues
			// with recursive calling of free_multi_stream by ~StreamAndEvent.
			while (!multi_streams.empty()) {
				free_multi_streams(multi_streams.begin()->first);
			}
		}

		std::unordered_map<hipStream_t, std::stack<std::shared_ptr<MultiStream>>> multi_streams;
	} stream_multi_streams;
	std::unordered_map<int, std::stack<std::shared_ptr<MultiStream>>> global_multi_streams;
};

StreamOwnedObjects& stream_owned_objects() {
	static StreamOwnedObjects s_stream_owned_objects;
	return s_stream_owned_objects;
}

std::unordered_map<hipStream_t, std::shared_ptr<GPUMemoryArena>>& stream_gpu_memory_arenas() {
	return stream_owned_objects().stream_gpu_memory_arenas;
}

std::unordered_map<int, std::shared_ptr<GPUMemoryArena>>& global_gpu_memory_arenas() {
	return stream_owned_objects().global_gpu_memory_arenas;
}

std::unordered_map<hipStream_t, std::stack<std::shared_ptr<MultiStream>>>& stream_multi_streams() {
	return stream_owned_objects().stream_multi_streams.multi_streams;
}

std::unordered_map<int, std::stack<std::shared_ptr<MultiStream>>>& global_multi_streams() {
	return stream_owned_objects().global_multi_streams;
}

TCNN_NAMESPACE_END
